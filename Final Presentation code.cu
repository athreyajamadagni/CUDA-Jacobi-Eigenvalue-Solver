#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <iostream>
const int num_mat = 3; // total number of matrices = total number of threads
const int N = 4;   // square symmetric matrix dimension
const int nTPB = 256;  // threads per block

// test symmetric matrices

  double a1[N*N] = {
      4.0,  -30.0,    60.0,   -35.0, 
    -30.0,  300.0,  -675.0,   420.0, 
     60.0, -675.0,  1620.0, -1050.0, 
    -35.0,  420.0, -1050.0,   700.0 };

  double a2[N*N] = {
    4.0, 0.0, 0.0, 0.0, 
    0.0, 1.0, 0.0, 0.0, 
    0.0, 0.0, 3.0, 0.0, 
    0.0, 0.0, 0.0, 2.0 };

  double a3[N*N] = {
    -2.0,   1.0,   0.0,   0.0,
     1.0,  -2.0,   1.0,   0.0,
     0.0,   1.0,  -2.0,   1.0,
     0.0,   0.0,   1.0,  -2.0 }; 


__host__ __device__
void r8mat_diag_get_vector ( int n, double a[], double v[] )
{
  int i;

  for ( i = 0; i < n; i++ )
  {
    v[i] = a[i+i*n];
  }

  return;
}

__host__ __device__
void r8mat_identity ( int n, double a[] )
{
  int i;
  int j;
  int k;

  k = 0;
  for ( j = 0; j < n; j++ )
  {
    for ( i = 0; i < n; i++ )
    {
      if ( i == j )
      {
        a[k] = 1.0;
      }
      else
      {
        a[k] = 0.0;
      }
      k = k + 1;
    }
  }

  return;
}

__host__ __device__
void jacobi_eigenvalue ( int n, double a[], int it_max, double v[], 
  double d[], int &it_num, int &rot_num )
{
  double *bw;
  double c;
  double g;
  double gapq;
  double h;
  int i;
  int j;
  int k;
  int l;
  int m;
  int p;
  int q;
  double s;
  double t;
  double tau;
  double term;
  double termp;
  double termq;
  double theta;
  double thresh;
  double w;
  double *zw;

  r8mat_identity ( n, v );

  r8mat_diag_get_vector ( n, a, d );

  bw = new double[n];
  zw = new double[n];

  for ( i = 0; i < n; i++ )
  {
    bw[i] = d[i];
    zw[i] = 0.0;
  }
  it_num = 0;
  rot_num = 0;

  while ( it_num < it_max )
  {
    it_num = it_num + 1;
//
//  The convergence threshold is based on the size of the elements in
//  the strict upper triangle of the matrix.
//
    thresh = 0.0;
    for ( j = 0; j < n; j++ )
    {
      for ( i = 0; i < j; i++ )
      {
        thresh = thresh + a[i+j*n] * a[i+j*n];
      }
    }

    thresh = sqrt ( thresh ) / ( double ) ( 4 * n );

    if ( thresh == 0.0 )
    {
      break;
    }

    for ( p = 0; p < n; p++ )
    {
      for ( q = p + 1; q < n; q++ )
      {
        gapq = 10.0 * fabs ( a[p+q*n] );
        termp = gapq + fabs ( d[p] );
        termq = gapq + fabs ( d[q] );
//
//  Annihilate tiny offdiagonal elements.
//
        if ( 4 < it_num &&
             termp == fabs ( d[p] ) &&
             termq == fabs ( d[q] ) )
        {
          a[p+q*n] = 0.0;
        }
//
//  Otherwise, apply a rotation.
//
        else if ( thresh <= fabs ( a[p+q*n] ) )
        {
          h = d[q] - d[p];
          term = fabs ( h ) + gapq;

          if ( term == fabs ( h ) )
          {
            t = a[p+q*n] / h;
          }
          else
          {
            theta = 0.5 * h / a[p+q*n];
            t = 1.0 / ( fabs ( theta ) + sqrt ( 1.0 + theta * theta ) );
            if ( theta < 0.0 )
            {
              t = - t;
            }
          }
          c = 1.0 / sqrt ( 1.0 + t * t );
          s = t * c;
          tau = s / ( 1.0 + c );
          h = t * a[p+q*n];
//
//  Accumulate corrections to diagonal elements.
//
          zw[p] = zw[p] - h;                 
          zw[q] = zw[q] + h;
          d[p] = d[p] - h;
          d[q] = d[q] + h;

          a[p+q*n] = 0.0;
//
//  Rotate, using information from the upper triangle of A only.
//
          for ( j = 0; j < p; j++ )
          {
            g = a[j+p*n];
            h = a[j+q*n];
            a[j+p*n] = g - s * ( h + g * tau );
            a[j+q*n] = h + s * ( g - h * tau );
          }

          for ( j = p + 1; j < q; j++ )
          {
            g = a[p+j*n];
            h = a[j+q*n];
            a[p+j*n] = g - s * ( h + g * tau );
            a[j+q*n] = h + s * ( g - h * tau );
          }

          for ( j = q + 1; j < n; j++ )
          {
            g = a[p+j*n];
            h = a[q+j*n];
            a[p+j*n] = g - s * ( h + g * tau );
            a[q+j*n] = h + s * ( g - h * tau );
          }
//
//  Accumulate information in the eigenvector matrix.
//
          for ( j = 0; j < n; j++ )
          {
            g = v[j+p*n];
            h = v[j+q*n];
            v[j+p*n] = g - s * ( h + g * tau );
            v[j+q*n] = h + s * ( g - h * tau );
          }
          rot_num = rot_num + 1;
        }
      }
    }

    for ( i = 0; i < n; i++ )
    {
      bw[i] = bw[i] + zw[i];
      d[i] = bw[i];
      zw[i] = 0.0;
    }
  }
//
//  Restore upper triangle of input matrix.
//
  for ( j = 0; j < n; j++ )
  {
    for ( i = 0; i < j; i++ )
    {
      a[i+j*n] = a[j+i*n];
    }
  }
//
//  Ascending sort the eigenvalues and eigenvectors.
//
  for ( k = 0; k < n - 1; k++ )
  {
    m = k;
    for ( l = k + 1; l < n; l++ )
    {
      if ( d[l] < d[m] )
      {
        m = l;
      }
    }

    if ( m != k )
    {
      t    = d[m];
      d[m] = d[k];
      d[k] = t;
      for ( i = 0; i < n; i++ )
      {
        w        = v[i+m*n];
        v[i+m*n] = v[i+k*n];
        v[i+k*n] = w;
      }
    }
  }

  delete [] bw;
  delete [] zw;

  return;
}


__global__ void je(int num_matr, int n, double *a, int it_max, double *v, double *d){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  int it_num;
  int rot_num;
  if (idx < num_matr){
    jacobi_eigenvalue(n, a+(idx*n*n), it_max, v+(idx*n*n), d+(idx*n), it_num, rot_num);
  }
}

void initialize_matrix(int mat_id, int n, double *mat, double *v){

  for (int i = 0; i < n*n; i++) *(v+(mat_id*n*n)+i) = mat[i];
}

void print_vec(int vec_id, int n, double *d){

  std::cout << "matrix " << vec_id << " eigenvalues: " << std::endl;
  for (int i = 0; i < n; i++) std::cout << i << ": " << *(d+(n*vec_id)+i) << std::endl;
  std::cout << std::endl;
}
int main(){
// make sure device heap has enough space for in-kernel new allocations
  const int heapsize = num_mat*N*sizeof(double)*2;
  const int chunks = heapsize/(8192*1024) + 1;
  hipError_t cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, (8192*1024) * chunks);
  if (cudaStatus != hipSuccess) {
        fprintf(stderr, "set device heap limit failed!");
    }
  const int max_iter = 1000;
  double *h_a, *d_a, *h_v, *d_v, *h_d, *d_d;
  h_a = (double *)malloc(num_mat*N*N*sizeof(double));
  h_v = (double *)malloc(num_mat*N*N*sizeof(double));
  h_d = (double *)malloc(num_mat*  N*sizeof(double));
  hipMalloc(&d_a, num_mat*N*N*sizeof(double));
  hipMalloc(&d_v, num_mat*N*N*sizeof(double));
  hipMalloc(&d_d, num_mat*  N*sizeof(double));
  memset(h_a, 0, num_mat*N*N*sizeof(double));
  memset(h_v, 0, num_mat*N*N*sizeof(double));
  memset(h_d, 0, num_mat*  N*sizeof(double));
  initialize_matrix(0, N, a1, h_a);
  initialize_matrix(1, N, a2, h_a);
  initialize_matrix(2, N, a3, h_a);
  hipMemcpy(d_a, h_a, num_mat*N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, num_mat*N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_d, h_d, num_mat*  N*sizeof(double), hipMemcpyHostToDevice);
  je<<<(num_mat+nTPB-1)/nTPB, nTPB>>>(num_mat, N, d_a, max_iter, d_v, d_d);
  hipMemcpy(h_d, d_d, num_mat*N*sizeof(double), hipMemcpyDeviceToHost);
  print_vec(0, N, h_d);
  print_vec(1, N, h_d);
  print_vec(2, N, h_d);
  return 0;
}